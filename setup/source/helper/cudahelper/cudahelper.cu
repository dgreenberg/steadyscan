#include "hip/hip_runtime.h"
#include "cudahelper.cuh"

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 256
#endif

template<class T>
bool CudaHelper<T>::initDevice( int device ) {
	hipError_t rc;
	void* ptr;
	hipDeviceProp_t prop;

	rc = hipGetDeviceProperties( &prop, device );
	if( rc != hipSuccess ) {
		std::cout << "unable to query device info for device " << device << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;
	}

	rc = hipSetDevice( device );
	if( rc != hipSuccess ) {
		std::cout << "unable to select " << prop.name << "(dev " << device << ")" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;
		return false;
	}

	//this initializes the device
	//which takes about 1 sec
	rc = hipMalloc( &ptr, 1 );
	if( rc != hipSuccess ) {
		std::cout << "unable to allocate any memory on device" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;
		return false;
	}

	hipFree( ptr );

	return true;
}

bool cudaCheckReturnCode( hipError_t rc, int line, const char* module ) {
	if( rc == hipError_t::hipSuccess ) return true;

	printf( "%s\nLine %i in %s\n", hipGetErrorName( rc ), line, module );
	return false;
}
bool cudaCheckReturnCode( hipsparseStatus_t rc, int line, const char* module ) {
	switch( rc ) {
	case HIPSPARSE_STATUS_SUCCESS:
		return true;
	case HIPSPARSE_STATUS_ALLOC_FAILED:
		printf( "HIPSPARSE_STATUS_ALLOC_FAILED\n" );
		break;
	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		printf( "HIPSPARSE_STATUS_ARCH_MISMATCH\n" );
		break;
	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		printf( "HIPSPARSE_STATUS_EXECUTION_FAILED\n" );
		break;
	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		printf( "HIPSPARSE_STATUS_INTERNAL_ERROR\n" );
		break;
	case HIPSPARSE_STATUS_INVALID_VALUE:
		printf( "HIPSPARSE_STATUS_INVALID_VALUE\n" );
		break;
	case HIPSPARSE_STATUS_MAPPING_ERROR:
		printf( "HIPSPARSE_STATUS_MAPPING_ERROR\n" );
		break;
	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		printf( "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED\n" );
		break;
	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		printf( "HIPSPARSE_STATUS_NOT_INITIALIZED\n" );
		break;
	case HIPSPARSE_STATUS_ZERO_PIVOT:
		printf( "HIPSPARSE_STATUS_ZERO_PIVOT\n" );
		break;
	default:
		printf( "CUSPARSE UNHANDLED ERROR\n" );
	}
	return false;
}
bool cudaCheckReturnCode( hipsolverStatus_t rc, int line, const char* module ) {
	switch( rc ) {
	case HIPSOLVER_STATUS_SUCCESS:
		return true;
	case HIPSOLVER_STATUS_ALLOC_FAILED:
		printf( "HIPSOLVER_STATUS_ALLOC_FAILED\n" );
		break;
	case HIPSOLVER_STATUS_ARCH_MISMATCH:
		printf( "HIPSOLVER_STATUS_ARCH_MISMATCH\n" );
		break;
	case HIPSOLVER_STATUS_EXECUTION_FAILED:
		printf( "HIPSOLVER_STATUS_EXECUTION_FAILED\n" );
		break;
	case HIPSOLVER_STATUS_INTERNAL_ERROR:
		printf( "HIPSOLVER_STATUS_INTERNAL_ERROR\n" );
		break;
	case CUSOLVER_STATUS_INVALID_LICENSE:
		printf( "CUSOLVER_STATUS_INVALID_LICENSE\n" );
		break;
	case HIPSOLVER_STATUS_INVALID_VALUE:
		printf( "HIPSOLVER_STATUS_INVALID_VALUE\n" );
		break;
	case HIPSOLVER_STATUS_MAPPING_ERROR:
		printf( "HIPSOLVER_STATUS_MAPPING_ERROR\n" );
		break;
	case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		printf( "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED\n" );
		break;
	case HIPSOLVER_STATUS_NOT_INITIALIZED:
		printf( "HIPSOLVER_STATUS_NOT_INITIALIZED\n" );
		break;
	case HIPSOLVER_STATUS_NOT_SUPPORTED:
		printf( "HIPSOLVER_STATUS_NOT_SUPPORTED\n" );
		break;
	case HIPSOLVER_STATUS_ZERO_PIVOT:
		printf( "HIPSOLVER_STATUS_ZERO_PIVOT\n" );
		break;
	}
	printf( "Line %i in %s\n", line, module);
	return false;
}


template<class T>
bool CudaHelper<T>::initCuSolver( DeviceMemory<T>& mem ) {
	//init cuSolver
	if( !CUCHECKRC( hipsolverSpCreate( &mem.cuSolverHandle ) ) ) {
		return false;
	}
	if( !CUCHECKRC( hipsparseCreateMatDescr( &mem.cuSparseDescription ) ) ) {
		return false;
	}
	if( !CUCHECKRC( hipsparseSetMatIndexBase( mem.cuSparseDescription, HIPSPARSE_INDEX_BASE_ZERO ) ) ) {
		return false;
	}
	if( !CUCHECKRC( hipsparseSetMatType( mem.cuSparseDescription, HIPSPARSE_MATRIX_TYPE_GENERAL ) ) ) {
		return false;
	}
	return true;
}
template<class T>
bool CudaHelper<T>::initCuSparse( DeviceMemory<T>& mem ) {
	//init suSparse
	if( !CUCHECKRC( hipsparseCreate( &mem.cuSparseHandle ) ) ) {
		return false;
	}

	return true;
}
template<class T>
void CudaHelper<T>::freeCuSolver( DeviceMemory<T>& mem ) {
	if( nullptr != mem.cuSolverHandle ) {
		CUCHECKRC( hipsolverSpDestroy( mem.cuSolverHandle ) );
	}

	if( nullptr != mem.cuSparseDescription ) {
		CUCHECKRC( hipsparseDestroyMatDescr( mem.cuSparseDescription ) );
	}
}

template<class T>
void CudaHelper<T>::freeCuSparse( DeviceMemory<T>& mem ) {
	if( nullptr != mem.cuSparseHandle ) {
		CUCHECKRC( hipsparseDestroy( mem.cuSparseHandle ) );
	}
}



template<class T>
__global__ void d_setArray( T* d_dest, int numElements, T value ) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if( idx >= numElements ) return;
	d_dest[idx] = value;
}

template<class T>
void CudaHelper<T>::setArray( T* d_dest, T value, int numElements, int threadsPerBlock ) {
	if( threadsPerBlock < 0 ) threadsPerBlock = THREADS_PER_BLOCK;

	int numBlocks = ( numElements + threadsPerBlock - 1 ) / threadsPerBlock;
	d_setArray<T><<< numBlocks, threadsPerBlock >>>( d_dest, numElements, value );
}

template<class T>
void CudaHelper<T>::printArray( T* src, int numElements ) {
	T* arr = new T[numElements];
	hipMemcpy( arr, src, sizeof( T ) * numElements, hipMemcpyDeviceToHost );

	for( int i = 0; i < numElements; i++ ) {
		printf( "[%i]: %f\n", i, static_cast<float>( arr[i] ) );
	}

	delete[] arr;
}

template<class T>
__global__ void cpykernel( T* d_dest, T* d_src, int numElements ) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if( idx >= numElements ) return;

	d_dest[idx] = d_src[idx];
}

template<class T>
void CudaHelper<T>::copy( T* d_dest, T* d_src, int numElements ) {
	int numBlocks = ( numElements + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK;
	cpykernel<T><<<numBlocks, THREADS_PER_BLOCK>>>( d_dest, d_src, numElements );
}


template<class T>
bool CudaHelper<T>::hAlloc( T** h_dest, size_t numElements, unsigned int flags, const char* name ) {
	hipError_t rc;

	rc = hipHostAlloc( h_dest, sizeof(T) * numElements, flags );
	if( rc != hipSuccess ) {
		std::cout << "unable to allocate host memory for '" << name << "'" << std::endl;
		std::cout << "tried to allocate " << sizeof(T) * numElements << " bytes" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;

		//allocate using default new
		*h_dest = new T[numElements];
		if( *h_dest == nullptr ){
			std::cout << "unable to allocate host memory for '" << name << "'" << std::endl;
			std::cout << "tried to allocate " << sizeof(T) * numElements << " bytes" << std::endl;
			return false;
		}

		//register memory
		rc = hipHostRegister( *h_dest, sizeof(T) * numElements, hipHostRegisterDefault );

		//continue as unpinned if not successfull
		if( rc != hipSuccess ) {
			std::cout << "unable to pin host memory for '" << name << "'" << std::endl;
			std::cout << "tried to pin " << sizeof(T) * numElements << " bytes" << std::endl;
			std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;
		}

		return true;
	}
	return true;
}
template<class T>
bool CudaHelper<T>::dAlloc( T** d_dest, size_t numElements, const char* name ) {
	hipError_t rc;

	rc = hipMalloc( d_dest, sizeof(T) * numElements );
	if( rc != hipSuccess ) {
		std::cout << "unable to allocate device memory for '" << name << "'" << std::endl;
		std::cout << "tried to allocate " << sizeof(T) * numElements << " bytes" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;

		return false;
	}

	return true;
}
template<class T>
bool CudaHelper<T>::dAlloc( T** d_dest, size_t numElements, const char* name, const T init ) {
	if( !dAlloc( d_dest, numElements, name ) ) return false;

	setArray( *d_dest, init, numElements );

	return true;
}
template<class T>
bool CudaHelper<T>::h2d( T** d_dest, NeutralMatrix<T, int, RowMajor<int>>& h_mat, const char* name ) {
	int numElements = h_mat.rows() * h_mat.cols();

	if( !dAlloc( d_dest, numElements, name ) ) return false;
	bool unregisterHost = true;
	bool retv = true;

	//try to pin host memory
	hipError_t rc = hipHostRegister( h_mat.data(), sizeof( T ) * numElements, hipHostRegisterDefault );

	//well if that doesn't work show warning and
	//make sure not to call hipHostUnregister
	if( rc != hipSuccess ) {
		std::cout << "unable to register host memory for '" << name << "'" << std::endl;
		std::cout << "tried to register " << sizeof( T ) * numElements << " bytes" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;

		unregisterHost = false;
	}

	//cuda host memory to device
	rc = hipMemcpy( *d_dest, h_mat.data(), sizeof( T ) * numElements, hipMemcpyHostToDevice );

	//if that fail show error and
	//release all allocated resources
	if( rc != hipSuccess ) {
		std::cout << "unable to copy host memory to device for '" << name << "'" << std::endl;
		std::cout << "tried to copy " << sizeof( T ) * numElements << " bytes" << std::endl;
		std::cout << "error info: " << rc << "> " << hipGetErrorName( rc ) << std::endl;

		zCudaFree( *d_dest );
		retv = false;
	}

	//unpin host memory if it's pinned
	if( unregisterHost ) hipHostUnregister( h_mat.data() );

	return retv;
}








template class CudaHelper<float>;
template class CudaHelper<double>;
template class CudaHelper<bool>;
template class CudaHelper<char>;
template class CudaHelper<short>;
template class CudaHelper<int>;
template class CudaHelper<long long>;

